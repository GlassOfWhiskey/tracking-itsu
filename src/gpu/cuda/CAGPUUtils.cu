#include "hip/hip_runtime.h"
/// \file CAGPUtils.cu
/// \brief
///
/// \author Iacopo Colonnelli, Politecnico di Torino
///
/// \copyright Copyright (C) 2017  Iacopo Colonnelli. \n\n
///   This program is free software: you can redistribute it and/or modify
///   it under the terms of the GNU General Public License as published by
///   the Free Software Foundation, either version 3 of the License, or
///   (at your option) any later version. \n\n
///   This program is distributed in the hope that it will be useful,
///   but WITHOUT ANY WARRANTY; without even the implied warranty of
///   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
///   GNU General Public License for more details. \n\n
///   You should have received a copy of the GNU General Public License
///   along with this program.  If not, see <http://www.gnu.org/licenses/>.
///////////////////////////////////////////////////////////////////////////////

#include "CAGPUUtils.h"

#include <sstream>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "CAGPUContext.h"
#include "CAMathUtils.h"

using namespace TRACKINGITSU_TARGET_NAMESPACE;

namespace {
void checkCUDAError(const hipError_t error, const char *file, const int line)
{
  if (error != hipSuccess) {

    std::ostringstream errorString { };

    errorString << file << ":" << line << " CUDA API returned error [" << hipGetErrorString(error) << "] (code "
        << error << ")" << std::endl;

    throw std::runtime_error { errorString.str() };
  }
}
}

dim3 CAGPUUtils::Host::getBlockSize(const int colsNum)
{
  return getBlockSize(colsNum, 1);
}

dim3 CAGPUUtils::Host::getBlockSize(const int colsNum, const int rowsNum)
{
  const CAGPUDeviceProperties& deviceProperties = CAGPUContext::getInstance().getDeviceProperties();
  int xThreads = min(colsNum, deviceProperties.maxThreadsDim.x);
  int yThreads = min(rowsNum, deviceProperties.maxThreadsDim.y);
  const int totalThreads = min(CAMathUtils::roundUp(xThreads * yThreads, deviceProperties.warpSize), deviceProperties.maxThreadsPerBlock);

  if (xThreads > yThreads) {

    xThreads = CAMathUtils::findNearestDivisor(xThreads, totalThreads);
    yThreads = totalThreads / xThreads;

  } else {

    yThreads = CAMathUtils::findNearestDivisor(yThreads, totalThreads);
    xThreads = totalThreads / yThreads;
  }

  return dim3 { static_cast<unsigned int>(xThreads), static_cast<unsigned int>(yThreads) };
}

void CAGPUUtils::Host::gpuMalloc(void **p, const int size)
{
  checkCUDAError(hipMalloc(p, size), __FILE__, __LINE__);
}

void CAGPUUtils::Host::gpuFree(void *p)
{
  checkCUDAError(hipFree(p), __FILE__, __LINE__);
}

void CAGPUUtils::Host::gpuMemset(void *p, int value, int size)
{
  checkCUDAError(hipMemset(p, value, size), __FILE__, __LINE__);
}

void CAGPUUtils::Host::gpuMemcpyHostToDevice(void *dst, const void *src, int size)
{
  checkCUDAError(hipMemcpy(dst, src, size, hipMemcpyHostToDevice), __FILE__, __LINE__);
}

void CAGPUUtils::Host::gpuMemcpyDeviceToHost(void *dst, const void *src, int size)
{
  checkCUDAError(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost), __FILE__, __LINE__);
}

void CAGPUUtils::Host::gpuStartProfiler()
{
  checkCUDAError(hipProfilerStart(), __FILE__, __LINE__);
}

void CAGPUUtils::Host::gpuStopProfiler()
{
  checkCUDAError(hipProfilerStop(), __FILE__, __LINE__);
}

GPU_DEVICE int CAGPUUtils::Device::getLaneIndex(const int warpSize)
{
  return (threadIdx.x + threadIdx.y * blockDim.x) % warpSize;
}

GPU_DEVICE int CAGPUUtils::Device::shareToWarp(const int value, const int laneIndex)
{
  return __shfl(value, laneIndex);
}

GPU_DEVICE int CAGPUUtils::Device::gpuAtomicAdd(int *p, const int incrementSize)
{
  return atomicAdd(p, incrementSize);
}
