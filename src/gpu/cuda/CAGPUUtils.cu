/// \file CAGPUtils.cu
/// \brief
///
/// \author Iacopo Colonnelli, Politecnico di Torino
///
/// \copyright Copyright (C) 2017  Iacopo Colonnelli. \n\n
///   This program is free software: you can redistribute it and/or modify
///   it under the terms of the GNU General Public License as published by
///   the Free Software Foundation, either version 3 of the License, or
///   (at your option) any later version. \n\n
///   This program is distributed in the hope that it will be useful,
///   but WITHOUT ANY WARRANTY; without even the implied warranty of
///   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
///   GNU General Public License for more details. \n\n
///   You should have received a copy of the GNU General Public License
///   along with this program.  If not, see <http://www.gnu.org/licenses/>.
///////////////////////////////////////////////////////////////////////////////

#include "CAGPUUtils.h"

#include <sstream>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace {
void checkCUDAError(const hipError_t error, const int line)
{
  if (error != hipSuccess) {

    std::ostringstream errorString{};

    errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << "), line(" << line
        << ")" << std::endl;

    throw std::runtime_error{ errorString.str() };
  }
}
}

void CAGPUUtils::gpuMalloc(void **p, const int size)
{
  checkCUDAError(hipMalloc(p, size), __LINE__);
}

void CAGPUUtils::gpuFree(void *p)
{
  checkCUDAError(hipFree(p), __LINE__);
}

void CAGPUUtils::gpuMemset(void *p, int value, int size)
{
  checkCUDAError(hipMemset(p, value, size), __LINE__);
}

void CAGPUUtils::gpuMemcpyHostToDevice(void *dst, const void *src, int size)
{
  checkCUDAError(hipMemcpy(dst, src, size, hipMemcpyHostToDevice), __LINE__);
}

void CAGPUUtils::gpuStartProfiler()
{
  checkCUDAError(hipProfilerStart(), __LINE__);
}

void CAGPUUtils::gpuStopProfiler()
{
  checkCUDAError(hipProfilerStop(), __LINE__);
}
