#include "hip/hip_runtime.h"
/// \file CAGPUTracker.cu
/// \brief
///
/// \author Iacopo Colonnelli, Politecnico di Torino
///
/// \copyright Copyright (C) 2017  Iacopo Colonnelli. \n\n
///   This program is free software: you can redistribute it and/or modify
///   it under the terms of the GNU General Public License as published by
///   the Free Software Foundation, either version 3 of the License, or
///   (at your option) any later version. \n\n
///   This program is distributed in the hope that it will be useful,
///   but WITHOUT ANY WARRANTY; without even the implied warranty of
///   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
///   GNU General Public License for more details. \n\n
///   You should have received a copy of the GNU General Public License
///   along with this program.  If not, see <http://www.gnu.org/licenses/>.
///////////////////////////////////////////////////////////////////////////////

#include "CATracker.h"

#include <array>
#include <sstream>
#include <iostream>

#include <hip/hip_runtime.h>

#include "CAConstants.h"
#include "CAGPUContext.h"
#include "CAGPUPrimaryVertexContext.h"
#include "CAGPUVector.h"
#include "CAIndexTableUtils.h"
#include "CAMathUtils.h"
#include "CAPrimaryVertexContext.h"
#include "CATrackingUtils.h"

__device__ void computeLayerTracklets(CAGPUPrimaryVertexContext& primaryVertexContext, const int layerIndex,
    int &clusterTracklets, const int warpSize, const bool dryRun)
{
  const int currentClusterIndex = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);
  const float3 &primaryVertex = primaryVertexContext.getPrimaryVertex();
  int startIndex;
  int currentIndex = 0;

  if (!dryRun) {

    int laneIndex = CAGPUUtils::Device::getLaneIndex(warpSize);

    if (laneIndex == warpSize - 1) {

      startIndex = primaryVertexContext.getTracklets()[layerIndex].extend(clusterTracklets);
    }

    startIndex = CAGPUUtils::Device::shareToWarp(startIndex, warpSize - 1);
    const int currentClusterOffset = __shfl_up(clusterTracklets, 1);

    if (laneIndex != 0) {

      startIndex += currentClusterOffset;
    }
  }

  if (currentClusterIndex < primaryVertexContext.getClusters()[layerIndex].size()) {

    const CACluster& currentCluster { primaryVertexContext.getClusters()[layerIndex][currentClusterIndex] };

    /*if (mUsedClustersTable[currentCluster.clusterId] != CAConstants::ITS::UnusedIndex) {

     continue;
     }*/

    const float tanLambda { (currentCluster.zCoordinate - primaryVertex.z) / currentCluster.rCoordinate };
    const float directionZIntersection { tanLambda
        * ((CAConstants::ITS::LayersRCoordinate())[layerIndex + 1] - currentCluster.rCoordinate)
        + currentCluster.zCoordinate };

    const GPU_ARRAY<int, 4> selectedBinsRect { CATrackingUtils::getBinsRect(currentCluster, layerIndex,
        directionZIntersection) };

    if (selectedBinsRect[0] != 0 || selectedBinsRect[1] != 0 || selectedBinsRect[2] != 0 || selectedBinsRect[3] != 0) {

      const int nextLayerClustersNum { static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size()) };
      int phiBinsNum { selectedBinsRect[3] - selectedBinsRect[1] + 1 };

      if (phiBinsNum < 0) {

        phiBinsNum += CAConstants::IndexTable::PhiBins;
      }

      if (!dryRun && layerIndex > 0) {

        primaryVertexContext.getTrackletsLookupTable()[layerIndex - 1][currentClusterIndex] = startIndex;
      }

      for (int iPhiBin { selectedBinsRect[1] }, iPhiCount { 0 }; iPhiCount < phiBinsNum;
          iPhiBin = ++iPhiBin == CAConstants::IndexTable::PhiBins ? 0 : iPhiBin, iPhiCount++) {

        const int firstBinIndex { CAIndexTableUtils::getBinIndex(selectedBinsRect[0], iPhiBin) };
        const int maxBinIndex { firstBinIndex + selectedBinsRect[2] - selectedBinsRect[0] + 1 };
        const int firstRowClusterIndex = primaryVertexContext.getIndexTables()[layerIndex][firstBinIndex];
        const int maxRowClusterIndex = primaryVertexContext.getIndexTables()[layerIndex][maxBinIndex];

        for (int iNextLayerCluster { firstRowClusterIndex };
            iNextLayerCluster <= maxRowClusterIndex && iNextLayerCluster < nextLayerClustersNum; ++iNextLayerCluster) {

          const CACluster& nextCluster { primaryVertexContext.getClusters()[layerIndex + 1][iNextLayerCluster] };

          if (CATrackingUtils::isValidTracklet(currentCluster, nextCluster, tanLambda, directionZIntersection)) {

            if (dryRun) {

              ++clusterTracklets;

            } else {

              primaryVertexContext.getTracklets()[layerIndex].emplace(startIndex + currentIndex, currentClusterIndex,
                  iNextLayerCluster, currentCluster, nextCluster);
              ++currentIndex;
            }
          }
        }
      }
    }
  }
}

__device__ void computeLayerCells(CAGPUPrimaryVertexContext& primaryVertexContext, const int layerIndex,
    int &trackletCells, const int warpSize, const bool dryRun)
{
  const int currentTrackletIndex = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);
  const float3 &primaryVertex = primaryVertexContext.getPrimaryVertex();
  int startIndex;
  int currentIndex = 0;

  if (!dryRun) {

    int laneIndex = CAGPUUtils::Device::getLaneIndex(warpSize);

    if (laneIndex == warpSize - 1) {

      startIndex = primaryVertexContext.getCells()[layerIndex].extend(trackletCells);
    }

    startIndex = CAGPUUtils::Device::shareToWarp(startIndex, warpSize - 1);
    const int currentTrackletOffset = __shfl_up(trackletCells, 1);

    if (laneIndex != 0) {

      startIndex += currentTrackletOffset;
    }
  }

  if (currentTrackletIndex < primaryVertexContext.getTracklets()[layerIndex].size()) {

    const CATracklet& currentTracklet { primaryVertexContext.getTracklets()[layerIndex][currentTrackletIndex] };
    const int nextLayerClusterIndex { currentTracklet.secondClusterIndex };
    const int nextLayerFirstTrackletIndex {
        primaryVertexContext.getTrackletsLookupTable()[layerIndex][nextLayerClusterIndex] };

    /*
     if (nextLayerFirstTrackletIndex == CAConstants::ITS::UnusedIndex) {

     continue;
     }
     */

    const CACluster& firstCellCluster {
        primaryVertexContext.getClusters()[layerIndex][currentTracklet.firstClusterIndex] };
    const CACluster& secondCellCluster {
        primaryVertexContext.getClusters()[layerIndex + 1][currentTracklet.secondClusterIndex] };
    const float firstCellClusterQuadraticRCoordinate { firstCellCluster.rCoordinate * firstCellCluster.rCoordinate };
    const float secondCellClusterQuadraticRCoordinate { secondCellCluster.rCoordinate * secondCellCluster.rCoordinate };
    const float3 firstDeltaVector { secondCellCluster.xCoordinate - firstCellCluster.xCoordinate,
        secondCellCluster.yCoordinate - firstCellCluster.yCoordinate, secondCellClusterQuadraticRCoordinate
            - firstCellClusterQuadraticRCoordinate };
    const int nextLayerTrackletsNum { static_cast<int>(primaryVertexContext.getTracklets()[layerIndex + 1].size()) };

    if (!dryRun && layerIndex > 0) {

      primaryVertexContext.getCellsLookupTable()[layerIndex - 1][currentTrackletIndex] = startIndex;
    }

    for (int iNextLayerTracklet { nextLayerFirstTrackletIndex };
        iNextLayerTracklet < nextLayerTrackletsNum
            && primaryVertexContext.getTracklets()[layerIndex + 1][iNextLayerTracklet].firstClusterIndex
                == nextLayerClusterIndex; ++iNextLayerTracklet) {

      const CATracklet& nextTracklet { primaryVertexContext.getTracklets()[layerIndex + 1][iNextLayerTracklet] };
      const float deltaTanLambda { MATH_ABS(currentTracklet.tanLambda - nextTracklet.tanLambda) };
      const float deltaPhi { MATH_ABS(currentTracklet.phiCoordinate - nextTracklet.phiCoordinate) };

      if (deltaTanLambda < CAConstants::Thresholds::CellMaxDeltaTanLambdaThreshold
          && (deltaPhi < CAConstants::Thresholds::CellMaxDeltaPhiThreshold
              || MATH_ABS(deltaPhi - CAConstants::Math::TwoPi) < CAConstants::Thresholds::CellMaxDeltaPhiThreshold)) {

        const float averageTanLambda { 0.5f * (currentTracklet.tanLambda + nextTracklet.tanLambda) };
        const float directionZIntersection { -averageTanLambda * firstCellCluster.rCoordinate
            + firstCellCluster.zCoordinate };
        const float deltaZ { MATH_ABS(directionZIntersection - primaryVertex.z) };

        if (deltaZ < CAConstants::Thresholds::CellMaxDeltaZThreshold()[layerIndex]) {

          const CACluster& thirdCellCluster {
              primaryVertexContext.getClusters()[layerIndex + 2][nextTracklet.secondClusterIndex] };

          const float thirdCellClusterQuadraticRCoordinate { thirdCellCluster.rCoordinate * thirdCellCluster.rCoordinate };

          const float3 secondDeltaVector { thirdCellCluster.xCoordinate - firstCellCluster.xCoordinate,
              thirdCellCluster.yCoordinate - firstCellCluster.yCoordinate, thirdCellClusterQuadraticRCoordinate
                  - firstCellClusterQuadraticRCoordinate };

          float3 cellPlaneNormalVector { CAMathUtils::crossProduct(firstDeltaVector, secondDeltaVector) };

          const float vectorNorm { std::sqrt(
              cellPlaneNormalVector.x * cellPlaneNormalVector.x + cellPlaneNormalVector.y * cellPlaneNormalVector.y
                  + cellPlaneNormalVector.z * cellPlaneNormalVector.z) };

          if (!(vectorNorm < CAConstants::Math::FloatMinThreshold
              || MATH_ABS(cellPlaneNormalVector.z) < CAConstants::Math::FloatMinThreshold)) {

            const float inverseVectorNorm { 1.0f / vectorNorm };
            const float3 normalizedPlaneVector { cellPlaneNormalVector.x * inverseVectorNorm, cellPlaneNormalVector.y
                * inverseVectorNorm, cellPlaneNormalVector.z * inverseVectorNorm };
            const float planeDistance { -normalizedPlaneVector.x * (secondCellCluster.xCoordinate - primaryVertex.x)
                - (normalizedPlaneVector.y * secondCellCluster.yCoordinate - primaryVertex.y)
                - normalizedPlaneVector.z * secondCellClusterQuadraticRCoordinate };
            const float normalizedPlaneVectorQuadraticZCoordinate { normalizedPlaneVector.z * normalizedPlaneVector.z };
            const float cellTrajectoryRadius { MATH_SQRT(
                (1.0f - normalizedPlaneVectorQuadraticZCoordinate - 4.0f * planeDistance * normalizedPlaneVector.z)
                    / (4.0f * normalizedPlaneVectorQuadraticZCoordinate)) };
            const float2 circleCenter { -0.5f * normalizedPlaneVector.x / normalizedPlaneVector.z, -0.5f
                * normalizedPlaneVector.y / normalizedPlaneVector.z };
            const float distanceOfClosestApproach { MATH_ABS(
                cellTrajectoryRadius - MATH_SQRT(circleCenter.x * circleCenter.x + circleCenter.y * circleCenter.y)) };

            if (distanceOfClosestApproach
                <= CAConstants::Thresholds::CellMaxDistanceOfClosestApproachThreshold()[layerIndex]) {

              if (dryRun) {

                ++trackletCells;

              } else {

                const float cellTrajectoryCurvature { 1.0f / cellTrajectoryRadius };

                primaryVertexContext.getCells()[layerIndex].emplace(startIndex + currentIndex,
                    currentTracklet.firstClusterIndex, nextTracklet.firstClusterIndex, nextTracklet.secondClusterIndex,
                    currentTrackletIndex, iNextLayerTracklet, normalizedPlaneVector, cellTrajectoryCurvature);
                ++currentIndex;
              }
            }
          }
        }
      }
    }
  }
}

__global__ void layerTrackletsKernel(CAGPUPrimaryVertexContext& primaryVertexContext, const int layerIndex, const int warpSize)
{
  int clusterTracklets = 0;
  const int laneIndex = CAGPUUtils::Device::getLaneIndex(warpSize);

  computeLayerTracklets(primaryVertexContext, layerIndex, clusterTracklets, warpSize, true);

  for (int iOffset = warpSize / 2; iOffset > 0; iOffset /= 2) {

    int clustersToSum = __shfl_up(clusterTracklets, iOffset);

    if (laneIndex >= iOffset) {

      clusterTracklets += clustersToSum;
    }
  }

  computeLayerTracklets(primaryVertexContext, layerIndex, clusterTracklets, warpSize, false);
}

__global__ void layerCellsKernel(CAGPUPrimaryVertexContext& primaryVertexContext, const int layerIndex, const int warpSize)
{
  int trackletCells = 0;
  const int laneIndex = CAGPUUtils::Device::getLaneIndex(warpSize);

  computeLayerCells(primaryVertexContext, layerIndex, trackletCells, warpSize, true);

  for (int iOffset = warpSize / 2; iOffset > 0; iOffset /= 2) {

    int trackletsToSum = __shfl_up(trackletCells, iOffset);

    if (laneIndex >= iOffset) {

      trackletCells += trackletsToSum;
    }
  }

  computeLayerCells(primaryVertexContext, layerIndex, trackletCells, warpSize, false);
}

template<>
void CATrackerTraits<true>::computeLayerTracklets(Context& primaryVertexContext, const int layerIndex)
{
  const CAGPUDeviceProperties& deviceProperties = CAGPUContext::getInstance().getDeviceProperties();
  const int clustersNum { static_cast<int>(primaryVertexContext.getClusters()[layerIndex].size()) };
  dim3 threadsPerBlock { CAGPUUtils::Host::getBlockSize(clustersNum) };
  dim3 blocksGrid { 1 + clustersNum / threadsPerBlock.x };

  hipStream_t currentStream;
  hipStreamCreate(&currentStream);

  layerTrackletsKernel<<< blocksGrid, threadsPerBlock, 0, currentStream >>>(primaryVertexContext.getDeviceContext(),
      layerIndex, deviceProperties.warpSize);

  hipError_t error = hipGetLastError();

  hipStreamDestroy(currentStream);

  if (error != hipSuccess) {

    std::ostringstream errorString { };
    errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << ")" << std::endl;

    throw std::runtime_error { errorString.str() };
  }
}

template<>
void CATrackerTraits<true>::postProcessTracklets(Context& primaryVertexContext)
{
  hipDeviceSynchronize();
}

template<>
void CATrackerTraits<true>::computeLayerCells(Context& primaryVertexContext, const int layerIndex)
{
  const CAGPUDeviceProperties& deviceProperties = CAGPUContext::getInstance().getDeviceProperties();
  const std::unique_ptr<int, void (*)(void*)> trackletsSizeUniquePointer =
      primaryVertexContext.getDeviceTracklets()[layerIndex].getSizeFromDevice();

  dim3 threadsPerBlock { CAGPUUtils::Host::getBlockSize(*trackletsSizeUniquePointer) };
  dim3 blocksGrid { 1 + *trackletsSizeUniquePointer / threadsPerBlock.x };

  hipStream_t currentStream;
  hipStreamCreate(&currentStream);

  layerCellsKernel<<< blocksGrid, threadsPerBlock, 0, currentStream >>>(primaryVertexContext.getDeviceContext(),
      layerIndex, deviceProperties.warpSize);

  hipStreamDestroy(currentStream);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    std::ostringstream errorString { };
    errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << ")" << std::endl;

    throw std::runtime_error { errorString.str() };
  }
}

template<>
void CATrackerTraits<true>::postProcessCells(Context& primaryVertexContext)
{
  hipDeviceSynchronize();

  for (int iLayer { 0 }; iLayer < CAConstants::ITS::CellsPerRoad; ++iLayer) {

    const std::unique_ptr<int, void (*)(void*)> cellsSizeUniquePointer =
        primaryVertexContext.getDeviceCells()[iLayer].getSizeFromDevice();
    primaryVertexContext.getDeviceCells()[iLayer].copyIntoVector(primaryVertexContext.getCells()[iLayer],
        *cellsSizeUniquePointer);

    if (iLayer < CAConstants::ITS::CellsPerRoad - 1) {

      const std::unique_ptr<int, void (*)(void*)> cellsLookupTableSizeUniquePointer =
          primaryVertexContext.getDeviceCellsLookupTable()[iLayer].getSizeFromDevice();
      primaryVertexContext.getDeviceCellsLookupTable()[iLayer].copyIntoVector(
          primaryVertexContext.getCellsLookupTable()[iLayer], *cellsLookupTableSizeUniquePointer);
    }
  }
}
