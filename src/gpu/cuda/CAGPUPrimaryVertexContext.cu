#include "hip/hip_runtime.h"
/// \file CAGPUPrimaryVertexContext.cxx
/// \brief
///
/// \author Iacopo Colonnelli, Politecnico di Torino
///
/// \copyright Copyright (C) 2017  Iacopo Colonnelli. \n\n
///   This program is free software: you can redistribute it and/or modify
///   it under the terms of the GNU General Public License as published by
///   the Free Software Foundation, either version 3 of the License, or
///   (at your option) any later version. \n\n
///   This program is distributed in the hope that it will be useful,
///   but WITHOUT ANY WARRANTY; without even the implied warranty of
///   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
///   GNU General Public License for more details. \n\n
///   You should have received a copy of the GNU General Public License
///   along with this program.  If not, see <http://www.gnu.org/licenses/>.
///////////////////////////////////////////////////////////////////////////////

#include "CAGPUPrimaryVertexContext.h"

namespace {
__global__ void fillIndexTables(CAGPUPrimaryVertexContext &primaryVertexContext)
{
  const int iLayer = threadIdx.x;

  const int layerClustersNum { static_cast<int>(primaryVertexContext.getClusters()[iLayer + 1].size()) };
  int previousBinIndex { 0 };
  primaryVertexContext.getIndexTables()[iLayer] = CAGPUArray<int,
      CAConstants::IndexTable::ZBins * CAConstants::IndexTable::PhiBins + 1> { };
  primaryVertexContext.getIndexTables()[iLayer][0] = 0;

  for (int iCluster { 0 }; iCluster < layerClustersNum; ++iCluster) {

    const int currentBinIndex { primaryVertexContext.getClusters()[iLayer + 1][iCluster].indexTableBinIndex };

    if (currentBinIndex > previousBinIndex) {

      for (int iBin { previousBinIndex + 1 }; iBin <= currentBinIndex; ++iBin) {

        primaryVertexContext.getIndexTables()[iLayer][iBin] = iCluster;
      }

      previousBinIndex = currentBinIndex;
    }
  }

  for (int iBin { previousBinIndex + 1 }; iBin <= CAConstants::IndexTable::ZBins * CAConstants::IndexTable::PhiBins;
      iBin++) {

    primaryVertexContext.getIndexTables()[iLayer][iBin] = layerClustersNum;
  }
}

__global__ void fillTrackletsLookupTables(CAGPUPrimaryVertexContext &primaryVertexContext)
{
  const int iLayer = threadIdx.x;
  const int tableSize = primaryVertexContext.getClusters()[iLayer + 1].size();

  for (int iBin = 0; iBin < tableSize; ++iBin) {

    primaryVertexContext.getTrackletsLookupTable()[iLayer][iBin] = CAConstants::ITS::UnusedIndex;
  }
}
}

CAGPUPrimaryVertexContext::CAGPUPrimaryVertexContext(const float3 &primaryVertex,
    const std::array<std::vector<CACluster>, CAConstants::ITS::LayersNumber> &clusters,
    const std::array<std::vector<CACell>, CAConstants::ITS::CellsPerRoad> &cells,
    const std::array<std::vector<int>, CAConstants::ITS::CellsPerRoad - 1> &cellsLookupTable)
    : mPrimaryVertex { primaryVertex }
{
  for (int iLayer { 0 }; iLayer < CAConstants::ITS::LayersNumber; ++iLayer) {

    this->mClusters[iLayer] =
        CAGPUVector<CACluster> { &clusters[iLayer][0], static_cast<int>(clusters[iLayer].size()) };

    if (iLayer < CAConstants::ITS::TrackletsPerRoad) {

      this->mTracklets[iLayer] = CAGPUVector<CATracklet> { static_cast<int>(std::ceil(
          (CAConstants::Memory::TrackletsMemoryCoefficients[iLayer] * clusters[iLayer].size())
              * clusters[iLayer + 1].size())) };
    }

    if (iLayer < CAConstants::ITS::CellsPerRoad) {

      this->mTrackletsLookupTable[iLayer] = CAGPUVector<int> { static_cast<int>(clusters[iLayer + 1].size()) };

      this->mCells[iLayer] = CAGPUVector<CACell> { static_cast<int>(cells[iLayer].capacity()) };
    }

    if (iLayer < CAConstants::ITS::CellsPerRoad - 1) {

      this->mCellsLookupTable[iLayer] = CAGPUVector<int> { &cellsLookupTable[iLayer][0],
          static_cast<int>(cellsLookupTable[iLayer].size()) };
    }
  }
}

GPU_DEVICE const float3& CAGPUPrimaryVertexContext::getPrimaryVertex()
{
  return *mPrimaryVertex;
}

GPU_HOST_DEVICE CAGPUArray<CAGPUVector<CACluster>, CAConstants::ITS::LayersNumber>& CAGPUPrimaryVertexContext::getClusters()
{
  return mClusters;
}

GPU_DEVICE CAGPUArray<CAGPUArray<int, CAConstants::IndexTable::ZBins * CAConstants::IndexTable::PhiBins + 1>,
    CAConstants::ITS::TrackletsPerRoad>& CAGPUPrimaryVertexContext::getIndexTables()
{
  return mIndexTables;
}

GPU_DEVICE CAGPUArray<CAGPUVector<CATracklet>, CAConstants::ITS::TrackletsPerRoad>& CAGPUPrimaryVertexContext::getTracklets()
{
  return mTracklets;
}

GPU_DEVICE CAGPUArray<CAGPUVector<int>, CAConstants::ITS::CellsPerRoad>& CAGPUPrimaryVertexContext::getTrackletsLookupTable()
{
  return mTrackletsLookupTable;
}

GPU_HOST_DEVICE CAGPUArray<CAGPUVector<CACell>, CAConstants::ITS::CellsPerRoad>& CAGPUPrimaryVertexContext::getCells()
{
  return mCells;
}

GPU_HOST_DEVICE CAGPUArray<CAGPUVector<int>, CAConstants::ITS::CellsPerRoad - 1>& CAGPUPrimaryVertexContext::getCellsLookupTable()
{
  return mCellsLookupTable;
}

CAPrimaryVertexContext<true>::CAPrimaryVertexContext(const CAEvent& event, const int primaryVertexIndex)
    : mPrimaryVertex { event.getPrimaryVertex(primaryVertexIndex) }, mClusters {
        CAPrimaryVertexContextInitializer<true>::initClusters(event, primaryVertexIndex) }, mCells {
        CAPrimaryVertexContextInitializer<true>::initCells(event) }, mCellsLookupTable {
        CAPrimaryVertexContextInitializer<true>::initCellsLookupTable(event) }, mGPUContext { mPrimaryVertex, mClusters, mCells,
        mCellsLookupTable }, mGPUContextDevicePointer { mGPUContext }
{
  fillIndexTables<<< 1, CAConstants::ITS::TrackletsPerRoad >>>(*mGPUContextDevicePointer);
  fillTrackletsLookupTables<<< 1, CAConstants::ITS::CellsPerRoad >>>(*mGPUContextDevicePointer);
  hipDeviceSynchronize();
}

const float3& CAPrimaryVertexContext<true>::getPrimaryVertex()
{
  return mPrimaryVertex;
}

std::array<std::vector<CACluster>, CAConstants::ITS::LayersNumber>& CAPrimaryVertexContext<true>::getClusters()
{
  return mClusters;
}

std::array<std::vector<CATracklet>, CAConstants::ITS::TrackletsPerRoad>& CAPrimaryVertexContext<true>::getTracklets()
{
  return mTracklets;
}

std::array<std::vector<int>, CAConstants::ITS::CellsPerRoad>& CAPrimaryVertexContext<true>::getTrackletsLookupTable()
{
  return mTrackletsLookupTable;
}

std::array<std::vector<CACell>, CAConstants::ITS::CellsPerRoad>& CAPrimaryVertexContext<true>::getCells()
{
  return mCells;
}

std::array<std::vector<int>, CAConstants::ITS::CellsPerRoad - 1>& CAPrimaryVertexContext<true>::getCellsLookupTable()
{
  return mCellsLookupTable;
}

std::array<std::vector<std::vector<int>>, CAConstants::ITS::CellsPerRoad - 1>& CAPrimaryVertexContext<true>::getCellsNeighbours()
{
  return mCellsNeighbours;
}

std::vector<CARoad>& CAPrimaryVertexContext<true>::getRoads()
{
  return mRoads;
}

CAGPUPrimaryVertexContext& CAPrimaryVertexContext<true>::getDeviceContext()
{
  return *mGPUContextDevicePointer;
}

CAGPUArray<CAGPUVector<CACluster>, CAConstants::ITS::LayersNumber>& CAPrimaryVertexContext<true>::getDeviceClusters()
{
  return mGPUContext.getClusters();
}

CAGPUArray<CAGPUVector<CATracklet>, CAConstants::ITS::TrackletsPerRoad>& CAPrimaryVertexContext<true>::getDeviceTracklets()
{
  return mGPUContext.getTracklets();
}

CAGPUArray<CAGPUVector<int>, CAConstants::ITS::CellsPerRoad>& CAPrimaryVertexContext<true>::getDeviceTrackletsLookupTable()
{
  return mGPUContext.getTrackletsLookupTable();
}

CAGPUArray<CAGPUVector<CACell>, CAConstants::ITS::CellsPerRoad>& CAPrimaryVertexContext<true>::getDeviceCells()
{
  return mGPUContext.getCells();
}

CAGPUArray<CAGPUVector<int>, CAConstants::ITS::CellsPerRoad - 1>& CAPrimaryVertexContext<true>::getDeviceCellsLookupTable()
{
  return mGPUContext.getCellsLookupTable();
}
