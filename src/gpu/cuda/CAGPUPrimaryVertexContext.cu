#include "hip/hip_runtime.h"
/// \file CAGPUPrimaryVertexContext.cxx
/// \brief
///
/// \author Iacopo Colonnelli, Politecnico di Torino
///
/// \copyright Copyright (C) 2017  Iacopo Colonnelli. \n\n
///   This program is free software: you can redistribute it and/or modify
///   it under the terms of the GNU General Public License as published by
///   the Free Software Foundation, either version 3 of the License, or
///   (at your option) any later version. \n\n
///   This program is distributed in the hope that it will be useful,
///   but WITHOUT ANY WARRANTY; without even the implied warranty of
///   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
///   GNU General Public License for more details. \n\n
///   You should have received a copy of the GNU General Public License
///   along with this program.  If not, see <http://www.gnu.org/licenses/>.
///////////////////////////////////////////////////////////////////////////////

#include "CAGPUPrimaryVertexContext.h"

#include <sstream>

#include "CAGPUStream.h"

namespace {
__device__ void fillIndexTables(CAGPUPrimaryVertexContext &primaryVertexContext, const int layerIndex)
{

  const int currentClusterIndex { static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x) };
  const int nextLayerClustersNum { static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size()) };

  if (currentClusterIndex < nextLayerClustersNum) {

    const int currentBinIndex {
        primaryVertexContext.getClusters()[layerIndex + 1][currentClusterIndex].indexTableBinIndex };
    int previousBinIndex;

    if (currentClusterIndex == 0) {

      primaryVertexContext.getIndexTables()[layerIndex][0] = 0;
      previousBinIndex = 0;

    } else {

      previousBinIndex = primaryVertexContext.getClusters()[layerIndex + 1][currentClusterIndex - 1].indexTableBinIndex;
    }

    if (currentBinIndex > previousBinIndex) {

      for (int iBin { previousBinIndex + 1 }; iBin <= currentBinIndex; ++iBin) {

        primaryVertexContext.getIndexTables()[layerIndex][iBin] = currentClusterIndex;
      }

      previousBinIndex = currentBinIndex;
    }

    if (currentClusterIndex == nextLayerClustersNum - 1) {

      for (int iBin { currentBinIndex + 1 }; iBin <= CAConstants::IndexTable::ZBins * CAConstants::IndexTable::PhiBins;
          iBin++) {

        primaryVertexContext.getIndexTables()[layerIndex][iBin] = nextLayerClustersNum;
      }
    }
  }
}

__device__ void fillTrackletsPerClusterTables(CAGPUPrimaryVertexContext &primaryVertexContext, const int layerIndex)
{
  const int currentClusterIndex { static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x) };
  const int clustersSize { static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size()) };

  if (currentClusterIndex < clustersSize) {

    primaryVertexContext.getTrackletsPerClusterTable()[layerIndex][currentClusterIndex] = 0;
  }
}

__device__ void fillCellsPerClusterTables(CAGPUPrimaryVertexContext &primaryVertexContext, const int layerIndex)
{
  const int totalThreadNum { static_cast<int>(primaryVertexContext.getClusters()[layerIndex + 1].size()) };
  const int trackletsSize { static_cast<int>(primaryVertexContext.getTracklets()[layerIndex + 1].capacity()) };
  const int trackletsPerThread { 1 + (trackletsSize - 1) / totalThreadNum };
  const int firstTrackletIndex { static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x) * trackletsPerThread };

  if (firstTrackletIndex < trackletsSize) {

    const int trackletsToSet { min(trackletsSize, firstTrackletIndex + trackletsPerThread) - firstTrackletIndex };
    memset(&primaryVertexContext.getCellsPerTrackletTable()[layerIndex][firstTrackletIndex], 0,
        trackletsToSet * sizeof(int));
  }
}

__global__ void fillDeviceStructures(CAGPUPrimaryVertexContext &primaryVertexContext, const int layerIndex)
{
  fillIndexTables(primaryVertexContext, layerIndex);

  if (layerIndex < CAConstants::ITS::CellsPerRoad) {

    fillTrackletsPerClusterTables(primaryVertexContext, layerIndex);
  }

  if (layerIndex < CAConstants::ITS::CellsPerRoad - 1) {

    fillCellsPerClusterTables(primaryVertexContext, layerIndex);
  }
}
}

CAGPUPrimaryVertexContext::CAGPUPrimaryVertexContext()
{
  // Nothing to do
}

CAGPUUniquePointer<CAGPUPrimaryVertexContext> CAGPUPrimaryVertexContext::initialize(const float3 &primaryVertex,
    const std::array<std::vector<CACluster>, CAConstants::ITS::LayersNumber> &clusters,
    const std::array<std::vector<CACell>, CAConstants::ITS::CellsPerRoad> &cells,
    const std::array<std::vector<int>, CAConstants::ITS::CellsPerRoad - 1> &cellsLookupTable)
{
  mPrimaryVertex = CAGPUUniquePointer<float3>{ primaryVertex };

  for (int iLayer { 0 }; iLayer < CAConstants::ITS::LayersNumber; ++iLayer) {

    this->mClusters[iLayer] =
        CAGPUVector<CACluster> { &clusters[iLayer][0], static_cast<int>(clusters[iLayer].size()) };

    if (iLayer < CAConstants::ITS::TrackletsPerRoad) {

      this->mTracklets[iLayer].reset(static_cast<int>(std::ceil(
          (CAConstants::Memory::TrackletsMemoryCoefficients[iLayer] * clusters[iLayer].size())
              * clusters[iLayer + 1].size())));
    }

    if (iLayer < CAConstants::ITS::CellsPerRoad) {

      this->mTrackletsLookupTable[iLayer].reset(static_cast<int>(clusters[iLayer + 1].size()));
      this->mTrackletsPerClusterTable[iLayer].reset(static_cast<int>(clusters[iLayer + 1].size()));
      this->mCells[iLayer].reset(static_cast<int>(cells[iLayer].capacity()));
    }

    if (iLayer < CAConstants::ITS::CellsPerRoad - 1) {

      this->mCellsLookupTable[iLayer].reset(static_cast<int>(cellsLookupTable[iLayer].size()));
      this->mCellsPerTrackletTable[iLayer].reset(static_cast<int>(cellsLookupTable[iLayer].size()));
    }
  }

  CAGPUUniquePointer<CAGPUPrimaryVertexContext> gpuContextDevicePointer { *this };

  std::array<CAGPUStream, CAConstants::ITS::LayersNumber> streamArray;

  for (int iLayer { 0 }; iLayer < CAConstants::ITS::TrackletsPerRoad; ++iLayer) {

    const int nextLayerClustersNum = static_cast<int>(clusters[iLayer + 1].size());

    dim3 threadsPerBlock { CAGPUUtils::Host::getBlockSize(nextLayerClustersNum) };
    dim3 blocksGrid { CAGPUUtils::Host::getBlocksGrid(threadsPerBlock, nextLayerClustersNum) };

    fillDeviceStructures<<< blocksGrid, threadsPerBlock, 0, streamArray[iLayer].get() >>>(*gpuContextDevicePointer, iLayer);

    hipError_t error = hipGetLastError();

    if (error != hipSuccess) {

      std::ostringstream errorString { };
      errorString << __FILE__ << ":" << __LINE__ << " CUDA API returned error [" << hipGetErrorString(error)
          << "] (code " << error << ")" << std::endl;

      throw std::runtime_error { errorString.str() };
    }
  }

  return gpuContextDevicePointer;
}
