#include "hip/hip_runtime.h"
/// \file CAGPUTrackingAPI.cu
/// \brief
///
/// \author Iacopo Colonnelli, Politecnico di Torino
///
/// \copyright Copyright (C) 2017  Iacopo Colonnelli. \n\n
///   This program is free software: you can redistribute it and/or modify
///   it under the terms of the GNU General Public License as published by
///   the Free Software Foundation, either version 3 of the License, or
///   (at your option) any later version. \n\n
///   This program is distributed in the hope that it will be useful,
///   but WITHOUT ANY WARRANTY; without even the implied warranty of
///   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
///   GNU General Public License for more details. \n\n
///   You should have received a copy of the GNU General Public License
///   along with this program.  If not, see <http://www.gnu.org/licenses/>.
///////////////////////////////////////////////////////////////////////////////

#include "CAGPUTrackingAPI.h"

#include <array>
#include <sstream>
#include <iostream>

#include <hip/hip_runtime.h>

#include "CAConstants.h"
#include "CAGPUVector.h"
#include "CAIndexTableUtils.h"
#include "CAMathUtils.h"
#include "CATrackingUtils.h"

namespace {
//TODO: this must be refined with runtime device queries or with careful planning
constexpr int WarpSize { 32 };
constexpr int MaxXThreads { 128 };
constexpr int MaxYThreads { 128 };
constexpr int MaxThreadsPerBlock { 128 };

hipStream_t streams[CAConstants::ITS::TrackletsPerRoad];

dim3 getBlockSize(const int colsNum, const int rowsNum)
{
  int xThreads = min(colsNum, MaxXThreads);
  int yThreads = min(rowsNum, MaxYThreads);
  const int totalThreads = min(CAMathUtils::roundUp(xThreads * yThreads, WarpSize), MaxThreadsPerBlock);

  if (xThreads > yThreads) {

    xThreads = CAMathUtils::findNearestDivisor(xThreads, totalThreads);
    yThreads = totalThreads / xThreads;

  } else {

    yThreads = CAMathUtils::findNearestDivisor(yThreads, totalThreads);
    xThreads = totalThreads / yThreads;
  }

  return dim3 { static_cast<unsigned int>(xThreads), static_cast<unsigned int>(yThreads) };
}

__device__ int getLaneIndex()
{
  return (threadIdx.x + threadIdx.y * blockDim.x) % WarpSize;
}

__device__ int shareToWarp(int value, int leaderIndex)
{
  return __shfl(value, leaderIndex);
}

__global__ void trackletsKernel(CAGPUPrimaryVertexContext *primaryVertexContext, const int layerIndex,
    const int currentClusterIndex, const float tanLambda, const float directionZIntersection, const int minZBinIndex,
    const int minPhiBinIndex, const int maxZBinIndex, const int phiBinsNum)
{
  int currentXIndex = static_cast<int>(blockDim.x * blockIdx.x + threadIdx.x);
  int currentYIndex = static_cast<int>(blockDim.y * blockIdx.y + threadIdx.y);

  __shared__ int rowFirstBinClusterIndexes[CAConstants::IndexTable::PhiBins];
  __shared__ int rowMaxBinClusterIndexes[CAConstants::IndexTable::PhiBins];

  if (currentYIndex < phiBinsNum) {

    const int phiBinIndex { (minPhiBinIndex + currentYIndex) % CAConstants::IndexTable::PhiBins };
    rowFirstBinClusterIndexes[phiBinIndex] = primaryVertexContext->indexTables[layerIndex][CAIndexTableUtils::getBinIndex(minZBinIndex,
        phiBinIndex)];
    rowMaxBinClusterIndexes[phiBinIndex] = primaryVertexContext->indexTables[layerIndex][CAIndexTableUtils::getBinIndex(maxZBinIndex + 1,
        phiBinIndex)];

    __syncthreads();

    const int nextClusterIndex { rowFirstBinClusterIndexes[phiBinIndex] + currentXIndex };

    if (nextClusterIndex <= rowMaxBinClusterIndexes[phiBinIndex]) {

      const CACluster& currentCluster { primaryVertexContext->clusters[layerIndex][currentClusterIndex] };
      const CACluster& nextCluster { primaryVertexContext->clusters[layerIndex + 1][nextClusterIndex] };

      if (CATrackingUtils::isValidTracklet(currentCluster, nextCluster, tanLambda, directionZIntersection)) {

        const int laneIndex { getLaneIndex() };
        const unsigned int mask { __ballot(1) };
        const int leaderIndex { __ffs(mask) - 1 };
        int startIndex { };

        if (laneIndex == leaderIndex) {

          startIndex = primaryVertexContext->tracklets[layerIndex].extend(__popc(mask));

          if (layerIndex > 0) {

            atomicMin(&primaryVertexContext->trackletsLookupTable[layerIndex - 1][currentClusterIndex], startIndex);
          }
        }

        startIndex = shareToWarp(startIndex, leaderIndex);

        primaryVertexContext->tracklets[layerIndex].insert(startIndex + __popc(mask & ((1 << laneIndex) - 1)), currentClusterIndex, nextClusterIndex,
            currentCluster, nextCluster);
      }
    }
  }
}
}

void CAGPUTrackingAPI::getTrackletsFromCluster(CAPrimaryVertexContext<true>& primaryVertexContext,
    const int currentLayerIndex, const int currentClusterIndex, const float tanLambda,
    const float directionZIntersection, const std::array<int, 4>& selectedBinsRect,
    const std::vector<std::pair<int, int>> &selectedClusters)
{

  const int rowsNum { static_cast<int>(selectedClusters.size()) };
  int maxClustersPerRow = 0;

  for (int iRow { 0 }; iRow < rowsNum; ++iRow) {

    if (selectedClusters[iRow].second > maxClustersPerRow) {

      maxClustersPerRow = selectedClusters[iRow].second;
    }
  }

  int phiBinsNum { selectedBinsRect[3] - selectedBinsRect[1] + 1 };
  if (phiBinsNum < 0) {

    phiBinsNum += CAConstants::IndexTable::PhiBins;
  }

  dim3 threadsPerBlock { getBlockSize(maxClustersPerRow, rowsNum) };
  dim3 blocksGrid { 1 + maxClustersPerRow / threadsPerBlock.x, 1 + rowsNum / threadsPerBlock.y };

  trackletsKernel<<< blocksGrid, threadsPerBlock, 0, streams[currentLayerIndex] >>>(primaryVertexContext.gpuContextDevicePointer,
      currentLayerIndex, currentClusterIndex, tanLambda, directionZIntersection,
      selectedBinsRect[0], selectedBinsRect[1], selectedBinsRect[2], phiBinsNum);

  hipError_t error = hipGetLastError();

  if (error != hipSuccess) {

    std::ostringstream errorString { };
    errorString << "CUDA API returned error [" << hipGetErrorString(error) << "] (code " << error << ")" << std::endl;

    throw std::runtime_error { errorString.str() };
  }
}
